#include "hip/hip_runtime.h"
/*
paper: 3D Gaussian Splatting for Real-Time Radiance Field Rendering, SIGGRAPH 2023
code:  https://github.com/graphdeco-inria/diff-gaussian-rasterization
*/
/*
 * Copyright (C) 2023, Inria
 * GRAPHDECO research group, https://team.inria.fr/graphdeco
 * All rights reserved.
 *
 * This software is free for non-commercial, research and evaluation use
 * under the terms of the LICENSE.md file.
 *
 * For inquiries contact  george.drettakis@inria.fr
 */

#include <hip/hip_cooperative_groups.h>
#include <cooperative_groups/reduce.h>

#include "gaussian_render.h"
#include "util.cuh"

namespace cg = cooperative_groups;

namespace GaussianRasterizer {
// Backward pass for conversion of spherical harmonics to RGB for
// each Gaussian.
__device__ void computeColorFromSH(int idx, int deg, int max_coeffs, const glm::vec3* means, glm::vec3 campos,
    const float* shs, const bool* clamped, const glm::vec3* dL_dcolor, glm::vec3* dL_dmeans, glm::vec3* dL_dshs) {
  // Compute intermediate values, as it is done during forward
  glm::vec3 pos      = means[idx];
  glm::vec3 dir_orig = pos - campos;
  glm::vec3 dir      = dir_orig / glm::length(dir_orig);

  glm::vec3* sh = ((glm::vec3*) shs) + idx * max_coeffs;

  // Use PyTorch rule for clamping: if clamping was applied,
  // gradient becomes 0.
  glm::vec3 dL_dRGB = dL_dcolor[idx];
  dL_dRGB.x *= clamped[3 * idx + 0] ? 0 : 1;
  dL_dRGB.y *= clamped[3 * idx + 1] ? 0 : 1;
  dL_dRGB.z *= clamped[3 * idx + 2] ? 0 : 1;

  glm::vec3 dRGBdx(0, 0, 0);
  glm::vec3 dRGBdy(0, 0, 0);
  glm::vec3 dRGBdz(0, 0, 0);
  float x = dir.x;
  float y = dir.y;
  float z = dir.z;

  // Target location for this Gaussian to write SH gradients to
  glm::vec3* dL_dsh = dL_dshs + idx * max_coeffs;

  // No tricks here, just high school-level calculus.
  float dRGBdsh0 = SH_C0;
  dL_dsh[0]      = dRGBdsh0 * dL_dRGB;
  if (deg > 0) {
    float dRGBdsh1 = -SH_C1 * y;
    float dRGBdsh2 = SH_C1 * z;
    float dRGBdsh3 = -SH_C1 * x;
    dL_dsh[1]      = dRGBdsh1 * dL_dRGB;
    dL_dsh[2]      = dRGBdsh2 * dL_dRGB;
    dL_dsh[3]      = dRGBdsh3 * dL_dRGB;

    dRGBdx = -SH_C1 * sh[3];
    dRGBdy = -SH_C1 * sh[1];
    dRGBdz = SH_C1 * sh[2];

    if (deg > 1) {
      float xx = x * x, yy = y * y, zz = z * z;
      float xy = x * y, yz = y * z, xz = x * z;

      float dRGBdsh4 = SH_C2[0] * xy;
      float dRGBdsh5 = SH_C2[1] * yz;
      float dRGBdsh6 = SH_C2[2] * (2.f * zz - xx - yy);
      float dRGBdsh7 = SH_C2[3] * xz;
      float dRGBdsh8 = SH_C2[4] * (xx - yy);
      dL_dsh[4]      = dRGBdsh4 * dL_dRGB;
      dL_dsh[5]      = dRGBdsh5 * dL_dRGB;
      dL_dsh[6]      = dRGBdsh6 * dL_dRGB;
      dL_dsh[7]      = dRGBdsh7 * dL_dRGB;
      dL_dsh[8]      = dRGBdsh8 * dL_dRGB;

      dRGBdx += SH_C2[0] * y * sh[4] + SH_C2[2] * 2.f * -x * sh[6] + SH_C2[3] * z * sh[7] + SH_C2[4] * 2.f * x * sh[8];
      dRGBdy += SH_C2[0] * x * sh[4] + SH_C2[1] * z * sh[5] + SH_C2[2] * 2.f * -y * sh[6] + SH_C2[4] * 2.f * -y * sh[8];
      dRGBdz += SH_C2[1] * y * sh[5] + SH_C2[2] * 2.f * 2.f * z * sh[6] + SH_C2[3] * x * sh[7];

      if (deg > 2) {
        float dRGBdsh9  = SH_C3[0] * y * (3.f * xx - yy);
        float dRGBdsh10 = SH_C3[1] * xy * z;
        float dRGBdsh11 = SH_C3[2] * y * (4.f * zz - xx - yy);
        float dRGBdsh12 = SH_C3[3] * z * (2.f * zz - 3.f * xx - 3.f * yy);
        float dRGBdsh13 = SH_C3[4] * x * (4.f * zz - xx - yy);
        float dRGBdsh14 = SH_C3[5] * z * (xx - yy);
        float dRGBdsh15 = SH_C3[6] * x * (xx - 3.f * yy);
        dL_dsh[9]       = dRGBdsh9 * dL_dRGB;
        dL_dsh[10]      = dRGBdsh10 * dL_dRGB;
        dL_dsh[11]      = dRGBdsh11 * dL_dRGB;
        dL_dsh[12]      = dRGBdsh12 * dL_dRGB;
        dL_dsh[13]      = dRGBdsh13 * dL_dRGB;
        dL_dsh[14]      = dRGBdsh14 * dL_dRGB;
        dL_dsh[15]      = dRGBdsh15 * dL_dRGB;

        dRGBdx += (SH_C3[0] * sh[9] * 3.f * 2.f * xy + SH_C3[1] * sh[10] * yz + SH_C3[2] * sh[11] * -2.f * xy +
                   SH_C3[3] * sh[12] * -3.f * 2.f * xz + SH_C3[4] * sh[13] * (-3.f * xx + 4.f * zz - yy) +
                   SH_C3[5] * sh[14] * 2.f * xz + SH_C3[6] * sh[15] * 3.f * (xx - yy));

        dRGBdy += (SH_C3[0] * sh[9] * 3.f * (xx - yy) + SH_C3[1] * sh[10] * xz +
                   SH_C3[2] * sh[11] * (-3.f * yy + 4.f * zz - xx) + SH_C3[3] * sh[12] * -3.f * 2.f * yz +
                   SH_C3[4] * sh[13] * -2.f * xy + SH_C3[5] * sh[14] * -2.f * yz + SH_C3[6] * sh[15] * -3.f * 2.f * xy);

        dRGBdz += (SH_C3[1] * sh[10] * xy + SH_C3[2] * sh[11] * 4.f * 2.f * yz +
                   SH_C3[3] * sh[12] * 3.f * (2.f * zz - xx - yy) + SH_C3[4] * sh[13] * 4.f * 2.f * xz +
                   SH_C3[5] * sh[14] * (xx - yy));
      }
    }
  }

  // The view direction is an input to the computation. View direction
  // is influenced by the Gaussian's mean, so SHs gradients
  // must propagate back into 3D position.
  glm::vec3 dL_ddir(glm::dot(dRGBdx, dL_dRGB), glm::dot(dRGBdy, dL_dRGB), glm::dot(dRGBdz, dL_dRGB));

  // Account for normalization of direction
  float3 dL_dmean = dnormvdv(float3{dir_orig.x, dir_orig.y, dir_orig.z}, float3{dL_ddir.x, dL_ddir.y, dL_ddir.z});

  // Gradients of loss w.r.t. Gaussian means, but only the portion
  // that is caused because the mean affects the view-dependent color.
  // Additional mean gradient is accumulated in below methods.
  dL_dmeans[idx] += glm::vec3(dL_dmean.x, dL_dmean.y, dL_dmean.z);
}

// Backward version of INVERSE 2D covariance matrix computation
// (due to length launched as separate kernel before other
// backward steps contained in preprocess)
__global__ void computeCov2DCUDA(int P, const float3* means, const int* radii, const float* cov3Ds, const float h_x,
    float h_y, const float tan_fovx, float tan_fovy, const float* view_matrix, const float* dL_dconics,
    float3* dL_dmeans, float* dL_dcov) {
  auto idx = cg::this_grid().thread_rank();
  if (idx >= P || !(radii[idx] > 0)) return;

  // Reading location of 3D covariance for this Gaussian
  const float* cov3D = cov3Ds + 6 * idx;

  // Fetch gradients, recompute 2D covariance and relevant
  // intermediate forward results needed in the backward.
  float3 mean      = means[idx];
  float3 dL_dconic = {dL_dconics[4 * idx], dL_dconics[4 * idx + 1], dL_dconics[4 * idx + 3]};
  float3 t         = transformPoint4x3(mean, view_matrix);

  const float limx = 1.3f * tan_fovx;
  const float limy = 1.3f * tan_fovy;
  const float txtz = t.x / t.z;
  const float tytz = t.y / t.z;
  t.x              = min(limx, max(-limx, txtz)) * t.z;
  t.y              = min(limy, max(-limy, tytz)) * t.z;

  const float x_grad_mul = txtz < -limx || txtz > limx ? 0 : 1;
  const float y_grad_mul = tytz < -limy || tytz > limy ? 0 : 1;

  glm::mat3 J =
      glm::mat3(h_x / t.z, 0.0f, -(h_x * t.x) / (t.z * t.z), 0.0f, h_y / t.z, -(h_y * t.y) / (t.z * t.z), 0, 0, 0);

  glm::mat3 W = glm::mat3(view_matrix[0], view_matrix[4], view_matrix[8], view_matrix[1], view_matrix[5],
      view_matrix[9], view_matrix[2], view_matrix[6], view_matrix[10]);
  // glm::mat3 W = glm::mat3(view_matrix[0], view_matrix[1], view_matrix[2], view_matrix[4], view_matrix[5],
  //     view_matrix[6], view_matrix[8], view_matrix[9], view_matrix[10]);

  glm::mat3 Vrk = glm::mat3(cov3D[0], cov3D[1], cov3D[2], cov3D[1], cov3D[3], cov3D[4], cov3D[2], cov3D[4], cov3D[5]);

  glm::mat3 T = W * J;

  glm::mat3 cov2D = glm::transpose(T) * glm::transpose(Vrk) * T;

  // Use helper variables for 2D covariance entries. More compact.
  float a = cov2D[0][0] += 0.3f;
  float b = cov2D[0][1];
  float c = cov2D[1][1] += 0.3f;

  float denom = a * c - b * b;
  float dL_da = 0, dL_db = 0, dL_dc = 0;
  float denom2inv = 1.0f / ((denom * denom) + 0.0000001f);

  if (denom2inv != 0) {
    // Gradients of loss w.r.t. entries of 2D covariance matrix,
    // given gradients of loss w.r.t. conic matrix (inverse covariance matrix).
    // e.g., dL / da = dL / d_conic_a * d_conic_a / d_a
    dL_da = denom2inv * (-c * c * dL_dconic.x + 2 * b * c * dL_dconic.y + (denom - a * c) * dL_dconic.z);
    dL_dc = denom2inv * (-a * a * dL_dconic.z + 2 * a * b * dL_dconic.y + (denom - a * c) * dL_dconic.x);
    dL_db = denom2inv * 2 * (b * c * dL_dconic.x - (denom + 2 * b * b) * dL_dconic.y + a * b * dL_dconic.z);

    // Gradients of loss L w.r.t. each 3D covariance matrix (Vrk) entry,
    // given gradients w.r.t. 2D covariance matrix (diagonal).
    // cov2D = transpose(T) * transpose(Vrk) * T;
    dL_dcov[6 * idx + 0] = (T[0][0] * T[0][0] * dL_da + T[0][0] * T[1][0] * dL_db + T[1][0] * T[1][0] * dL_dc);
    dL_dcov[6 * idx + 3] = (T[0][1] * T[0][1] * dL_da + T[0][1] * T[1][1] * dL_db + T[1][1] * T[1][1] * dL_dc);
    dL_dcov[6 * idx + 5] = (T[0][2] * T[0][2] * dL_da + T[0][2] * T[1][2] * dL_db + T[1][2] * T[1][2] * dL_dc);

    // Gradients of loss L w.r.t. each 3D covariance matrix (Vrk) entry,
    // given gradients w.r.t. 2D covariance matrix (off-diagonal).
    // Off-diagonal elements appear twice --> double the gradient.
    // cov2D = transpose(T) * transpose(Vrk) * T;
    dL_dcov[6 * idx + 1] =
        2 * T[0][0] * T[0][1] * dL_da + (T[0][0] * T[1][1] + T[0][1] * T[1][0]) * dL_db + 2 * T[1][0] * T[1][1] * dL_dc;
    dL_dcov[6 * idx + 2] =
        2 * T[0][0] * T[0][2] * dL_da + (T[0][0] * T[1][2] + T[0][2] * T[1][0]) * dL_db + 2 * T[1][0] * T[1][2] * dL_dc;
    dL_dcov[6 * idx + 4] =
        2 * T[0][2] * T[0][1] * dL_da + (T[0][1] * T[1][2] + T[0][2] * T[1][1]) * dL_db + 2 * T[1][1] * T[1][2] * dL_dc;
  } else {
    for (int i = 0; i < 6; i++) dL_dcov[6 * idx + i] = 0;
  }

  // Gradients of loss w.r.t. upper 2x3 portion of intermediate matrix T
  // cov2D = transpose(T) * transpose(Vrk) * T;
  float dL_dT00 = 2 * (T[0][0] * Vrk[0][0] + T[0][1] * Vrk[0][1] + T[0][2] * Vrk[0][2]) * dL_da +
                  (T[1][0] * Vrk[0][0] + T[1][1] * Vrk[0][1] + T[1][2] * Vrk[0][2]) * dL_db;
  float dL_dT01 = 2 * (T[0][0] * Vrk[1][0] + T[0][1] * Vrk[1][1] + T[0][2] * Vrk[1][2]) * dL_da +
                  (T[1][0] * Vrk[1][0] + T[1][1] * Vrk[1][1] + T[1][2] * Vrk[1][2]) * dL_db;
  float dL_dT02 = 2 * (T[0][0] * Vrk[2][0] + T[0][1] * Vrk[2][1] + T[0][2] * Vrk[2][2]) * dL_da +
                  (T[1][0] * Vrk[2][0] + T[1][1] * Vrk[2][1] + T[1][2] * Vrk[2][2]) * dL_db;
  float dL_dT10 = 2 * (T[1][0] * Vrk[0][0] + T[1][1] * Vrk[0][1] + T[1][2] * Vrk[0][2]) * dL_dc +
                  (T[0][0] * Vrk[0][0] + T[0][1] * Vrk[0][1] + T[0][2] * Vrk[0][2]) * dL_db;
  float dL_dT11 = 2 * (T[1][0] * Vrk[1][0] + T[1][1] * Vrk[1][1] + T[1][2] * Vrk[1][2]) * dL_dc +
                  (T[0][0] * Vrk[1][0] + T[0][1] * Vrk[1][1] + T[0][2] * Vrk[1][2]) * dL_db;
  float dL_dT12 = 2 * (T[1][0] * Vrk[2][0] + T[1][1] * Vrk[2][1] + T[1][2] * Vrk[2][2]) * dL_dc +
                  (T[0][0] * Vrk[2][0] + T[0][1] * Vrk[2][1] + T[0][2] * Vrk[2][2]) * dL_db;

  // Gradients of loss w.r.t. upper 3x2 non-zero entries of Jacobian matrix
  // T = W * J
  float dL_dJ00 = W[0][0] * dL_dT00 + W[0][1] * dL_dT01 + W[0][2] * dL_dT02;
  float dL_dJ02 = W[2][0] * dL_dT00 + W[2][1] * dL_dT01 + W[2][2] * dL_dT02;
  float dL_dJ11 = W[1][0] * dL_dT10 + W[1][1] * dL_dT11 + W[1][2] * dL_dT12;
  float dL_dJ12 = W[2][0] * dL_dT10 + W[2][1] * dL_dT11 + W[2][2] * dL_dT12;

  float tz  = 1.f / t.z;
  float tz2 = tz * tz;
  float tz3 = tz2 * tz;

  // Gradients of loss w.r.t. transformed Gaussian mean t
  float dL_dtx = x_grad_mul * -h_x * tz2 * dL_dJ02;
  float dL_dty = y_grad_mul * -h_y * tz2 * dL_dJ12;
  float dL_dtz =
      -h_x * tz2 * dL_dJ00 - h_y * tz2 * dL_dJ11 + (2 * h_x * t.x) * tz3 * dL_dJ02 + (2 * h_y * t.y) * tz3 * dL_dJ12;

  // Account for transformation of mean to t
  // t = transformPoint4x3(mean, view_matrix);
  float3 dL_dmean = transformVec4x3Transpose({dL_dtx, dL_dty, dL_dtz}, view_matrix);

  // Gradients of loss w.r.t. Gaussian means, but only the portion
  // that is caused because the mean affects the covariance matrix.
  // Additional mean gradient is accumulated in BACKWARD::preprocess.
  dL_dmeans[idx] = dL_dmean;
}

// Backward pass for the conversion of scale and rotation to a
// 3D covariance matrix for each Gaussian.
__device__ void computeCov3D(int idx, const glm::vec3 scale, float mod, const glm::vec4 rot, const float* dL_dcov3Ds,
    glm::vec3* dL_dscales, glm::vec4* dL_drots) {
  // Recompute (intermediate) results for the 3D covariance computation.
  glm::vec4 q = rot;  // / glm::length(rot);
  float r     = q.x;
  float x     = q.y;
  float y     = q.z;
  float z     = q.w;

  glm::mat3 R = glm::mat3(1.f - 2.f * (y * y + z * z), 2.f * (x * y - r * z), 2.f * (x * z + r * y),
      2.f * (x * y + r * z), 1.f - 2.f * (x * x + z * z), 2.f * (y * z - r * x), 2.f * (x * z - r * y),
      2.f * (y * z + r * x), 1.f - 2.f * (x * x + y * y));

  glm::mat3 S = glm::mat3(1.0f);

  glm::vec3 s = mod * scale;
  S[0][0]     = s.x;
  S[1][1]     = s.y;
  S[2][2]     = s.z;

  glm::mat3 M = S * R;

  const float* dL_dcov3D = dL_dcov3Ds + 6 * idx;

  glm::vec3 dunc(dL_dcov3D[0], dL_dcov3D[3], dL_dcov3D[5]);
  glm::vec3 ounc = 0.5f * glm::vec3(dL_dcov3D[1], dL_dcov3D[2], dL_dcov3D[4]);

  // Convert per-element covariance loss gradients to matrix form
  glm::mat3 dL_dSigma = glm::mat3(dL_dcov3D[0], 0.5f * dL_dcov3D[1], 0.5f * dL_dcov3D[2], 0.5f * dL_dcov3D[1],
      dL_dcov3D[3], 0.5f * dL_dcov3D[4], 0.5f * dL_dcov3D[2], 0.5f * dL_dcov3D[4], dL_dcov3D[5]);

  // Compute loss gradient w.r.t. matrix M
  // dSigma_dM = 2 * M
  glm::mat3 dL_dM = 2.0f * M * dL_dSigma;

  glm::mat3 Rt     = glm::transpose(R);
  glm::mat3 dL_dMt = glm::transpose(dL_dM);

  // Gradients of loss w.r.t. scale
  glm::vec3* dL_dscale = dL_dscales + idx;
  dL_dscale->x         = glm::dot(Rt[0], dL_dMt[0]);
  dL_dscale->y         = glm::dot(Rt[1], dL_dMt[1]);
  dL_dscale->z         = glm::dot(Rt[2], dL_dMt[2]);

  dL_dMt[0] *= s.x;
  dL_dMt[1] *= s.y;
  dL_dMt[2] *= s.z;

  // Gradients of loss w.r.t. normalized quaternion
  glm::vec4 dL_dq;
  dL_dq.x = 2 * z * (dL_dMt[0][1] - dL_dMt[1][0]) + 2 * y * (dL_dMt[2][0] - dL_dMt[0][2]) +
            2 * x * (dL_dMt[1][2] - dL_dMt[2][1]);
  dL_dq.y = 2 * y * (dL_dMt[1][0] + dL_dMt[0][1]) + 2 * z * (dL_dMt[2][0] + dL_dMt[0][2]) +
            2 * r * (dL_dMt[1][2] - dL_dMt[2][1]) - 4 * x * (dL_dMt[2][2] + dL_dMt[1][1]);
  dL_dq.z = 2 * x * (dL_dMt[1][0] + dL_dMt[0][1]) + 2 * r * (dL_dMt[2][0] - dL_dMt[0][2]) +
            2 * z * (dL_dMt[1][2] + dL_dMt[2][1]) - 4 * y * (dL_dMt[2][2] + dL_dMt[0][0]);
  dL_dq.w = 2 * r * (dL_dMt[0][1] - dL_dMt[1][0]) + 2 * x * (dL_dMt[2][0] + dL_dMt[0][2]) +
            2 * y * (dL_dMt[1][2] + dL_dMt[2][1]) - 4 * z * (dL_dMt[1][1] + dL_dMt[0][0]);

  // Gradients of loss w.r.t. unnormalized quaternion
  float4* dL_drot = (float4*) (dL_drots + idx);
  *dL_drot        = float4{dL_dq.x, dL_dq.y, dL_dq.z, dL_dq.w};
  // dnormvdv(float4{ rot.x, rot.y, rot.z, rot.w }, float4{ dL_dq.x, dL_dq.y, dL_dq.z, dL_dq.w });
}

// Backward pass of the preprocessing steps, except
// for the covariance computation and inversion
// (those are handled by a previous kernel call)
template <int C>
__global__ void preprocessCUDA_backward(int P, int D, int M, const float3* means, const int* radii, const float* shs,
    const bool* clamped, const glm::vec3* scales, const glm::vec4* rotations, const float scale_modifier,
    const float* proj, const glm::vec3* campos, const float3* dL_dmean2D, glm::vec3* dL_dmeans, float* dL_dcolor,
    float* dL_dcov3D, float* dL_dsh, glm::vec3* dL_dscale, glm::vec4* dL_drot) {
  auto idx = cg::this_grid().thread_rank();
  if (idx >= P || !(radii[idx] > 0)) return;

  float3 m = means[idx];

  // Taking care of gradients from the screenspace points
  float4 m_hom = transformPoint4x4(m, proj);
  float m_w    = 1.0f / (m_hom.w + 0.0000001f);

  // Compute loss gradient w.r.t. 3D means due to gradients of 2D means
  // from rendering procedure
  glm::vec3 dL_dmean;
  float mul1 = (proj[0] * m.x + proj[4] * m.y + proj[8] * m.z + proj[12]) * m_w * m_w;
  float mul2 = (proj[1] * m.x + proj[5] * m.y + proj[9] * m.z + proj[13]) * m_w * m_w;
  dL_dmean.x =
      (proj[0] * m_w - proj[3] * mul1) * dL_dmean2D[idx].x + (proj[1] * m_w - proj[3] * mul2) * dL_dmean2D[idx].y;
  dL_dmean.y =
      (proj[4] * m_w - proj[7] * mul1) * dL_dmean2D[idx].x + (proj[5] * m_w - proj[7] * mul2) * dL_dmean2D[idx].y;
  dL_dmean.z =
      (proj[8] * m_w - proj[11] * mul1) * dL_dmean2D[idx].x + (proj[9] * m_w - proj[11] * mul2) * dL_dmean2D[idx].y;

  // That's the second part of the mean gradient. Previous computation
  // of cov2D and following SH conversion also affects it.
  dL_dmeans[idx] += dL_dmean;

  // Compute gradient updates due to computing colors from SHs
  if (shs)
    computeColorFromSH(idx, D, M, (glm::vec3*) means, *campos, shs, clamped, (glm::vec3*) dL_dcolor,
        (glm::vec3*) dL_dmeans, (glm::vec3*) dL_dsh);

  // Compute gradient updates due to computing covariance from scale/rotation
  if (scales) computeCov3D(idx, scales[idx], scale_modifier, rotations[idx], dL_dcov3D, dL_dscale, dL_drot);
}

void preprocess_backward(int P, int D, int M, const float3* means3D, const int* radii, const float* shs,
    const bool* clamped, const glm::vec3* scales, const glm::vec4* rotations, const float scale_modifier,
    const float* cov3Ds, const float* viewmatrix, const float* projmatrix, const float focal_x, float focal_y,
    const float tan_fovx, float tan_fovy, const glm::vec3* campos, const float3* dL_dmean2D, const float* dL_dconic,
    glm::vec3* dL_dmean3D, float* dL_dcolor, float* dL_dcov3D, float* dL_dsh, glm::vec3* dL_dscale,
    glm::vec4* dL_drot) {
  // Propagate gradients for the path of 2D conic matrix computation.
  // Somewhat long, thus it is its own kernel rather than being part of
  // "preprocess". When done, loss gradient w.r.t. 3D means has been
  // modified and gradient w.r.t. 3D covariance matrix has been computed.
  computeCov2DCUDA KERNEL_ARG((P + 255) / 256, 256)(P, means3D, radii, cov3Ds, focal_x, focal_y, tan_fovx, tan_fovy,
      viewmatrix, dL_dconic, (float3*) dL_dmean3D, dL_dcov3D);

  // Propagate gradients for remaining steps: finish 3D mean gradients,
  // propagate color gradients to SH (if desireD), propagate 3D covariance
  // matrix gradients to scale and rotation.
  preprocessCUDA_backward<NUM_CHANNELS> KERNEL_ARG((P + 255) / 256, 256)(P, D, M, (float3*) means3D, radii, shs,
      clamped, (glm::vec3*) scales, (glm::vec4*) rotations, scale_modifier, projmatrix, campos, (float3*) dL_dmean2D,
      (glm::vec3*) dL_dmean3D, dL_dcolor, dL_dcov3D, dL_dsh, dL_dscale, dL_drot);
}

void render_backward(const dim3 grid, const dim3 block, const uint2* ranges, const uint32_t* point_list, int W, int H,
    int E, /*const float* bg_color,*/ const float2* means2D, const float4* conic_opacity, const float* colors,
    const float* extras, const float* out_opacity, const uint32_t* n_contrib, const float* dL_dpixels,
    const float* dL_dout_extras, const float* dL_dout_opacity, float3* dL_dmean2D, float4* dL_dconic2D,
    float* dL_dopacity, float* dL_dcolors, float* dL_dextras);

// Produce necessary gradients for optimization, corresponding to forward render pass
void Rasterizer::backward(const int P, int D, int M, int R, int E, const int width, int height, const float* means3D,
    const float* shs, const float* colors_precomp, const float* scales, const float scale_modifier,
    const float* rotations, const float* cov3D_precomp, const float* viewmatrix, const float* projmatrix,
    const float* campos, const float* extra, const float tan_fovx, float tan_fovy, const int* radii, char* geom_buffer,
    char* binning_buffer, char* img_buffer, const float* out_opacity, const float* dL_dpix,
    const float* dL_dout_opacity, const float* dL_dout_extra, float* dL_dmean2D, float* dL_dconic, float* dL_dopacity,
    float* dL_dcolor, float* dL_dmean3D, float* dL_dcov3D, float* dL_dsh, float* dL_dscale, float* dL_drot,
    float* dL_dextra, bool debug) {
  GeometryState geomState   = GeometryState::fromChunk(geom_buffer, P);
  BinningState binningState = BinningState::fromChunk(binning_buffer, R);
  ImageState imgState       = ImageState::fromChunk(img_buffer, width * height);

  if (radii == nullptr) {
    radii = geomState.internal_radii;
  }

  const float focal_y = height / (2.0f * tan_fovy);
  const float focal_x = width / (2.0f * tan_fovx);

  const dim3 tile_grid((width + BLOCK_X - 1) / BLOCK_X, (height + BLOCK_Y - 1) / BLOCK_Y, 1);
  const dim3 block(BLOCK_X, BLOCK_Y, 1);

  // Compute loss gradients w.r.t. 2D mean position, conic matrix,
  // opacity and RGB of Gaussians from per-pixel loss gradients.
  // If we were given precomputed colors and not SHs, use them.
  const float* color_ptr = (colors_precomp != nullptr) ? colors_precomp : geomState.rgb;
  render_backward(tile_grid, block, imgState.ranges, binningState.point_list, width, height, E, geomState.means2D,
      geomState.conic_opacity, color_ptr, extra, out_opacity, imgState.n_contrib, dL_dpix, dL_dout_extra,
      dL_dout_opacity, (float3*) dL_dmean2D, (float4*) dL_dconic, dL_dopacity, dL_dcolor, dL_dextra);
  CHECK_CUDA_ERROR("render");

  // Take care of the rest of preprocessing. Was the precomputed covariance
  // given to us or a scales/rot pair? If precomputed, pass that. If not,
  // use the one we computed ourselves.
  const float* cov3D_ptr = (cov3D_precomp != nullptr) ? cov3D_precomp : geomState.cov3D;
  preprocess_backward(P, D, M, (float3*) means3D, radii, shs, geomState.clamped, (glm::vec3*) scales,
      (glm::vec4*) rotations, scale_modifier, cov3D_ptr, viewmatrix, projmatrix, focal_x, focal_y, tan_fovx, tan_fovy,
      (glm::vec3*) campos, (float3*) dL_dmean2D, dL_dconic, (glm::vec3*) dL_dmean3D, dL_dcolor, dL_dcov3D, dL_dsh,
      (glm::vec3*) dL_dscale, (glm::vec4*) dL_drot);
  if (debug) hipDeviceSynchronize();
  CHECK_CUDA_ERROR("preprocess");
}

std::tuple<Tensor, Tensor, Tensor, Tensor, Tensor, Tensor, Tensor, Tensor, at::optional<Tensor>>
    RasterizeGaussiansBackwardCUDA(
        // scalar parameters
        const float scale_modifier, const float tan_fovx, const float tan_fovy, const int degree, const bool debug,
        // tensor parameters
        const torch::Tensor& viewmatrix, const torch::Tensor& projmatrix, const torch::Tensor& campos,
        // inputs
        const torch::Tensor& means3D, const torch::Tensor& colors, const at::optional<Tensor> extras,
        const torch::Tensor& scales, const torch::Tensor& rotations, const torch::Tensor& cov3D_precomp,
        const torch::Tensor& sh,
        // outputs
        const int R, const torch::Tensor& radii, const Tensor& out_opacity,
        // grad_outputs
        const torch::Tensor& dL_dout_color, const Tensor& dL_dout_opacity, const at::optional<Tensor>& dL_dout_extra,
        // grad_inputs
        torch::optional<Tensor>& grad_means2D, torch::optional<Tensor>& grad_conic,
        torch::optional<Tensor>& grad_opacity,
        // buffer
        const torch::Tensor& geomBuffer, const torch::Tensor& binningBuffer, const torch::Tensor& imageBuffer) {
  const int P = means3D.size(0);
  const int H = dL_dout_color.size(1);
  const int W = dL_dout_color.size(2);
  const int E = (extras.has_value() && dL_dout_extra.has_value()) ? extras.value().size(-1) : 0;

  int M = 0;
  if (sh.size(0) != 0) {
    M = sh.size(1);
  }

  Tensor dL_dmeans3D   = torch::zeros({P, 3}, means3D.options());
  Tensor dL_dmeans2D   = grad_means2D.has_value() ? grad_means2D.value() : torch::zeros({P, 3}, means3D.options());
  Tensor dL_dconic     = grad_conic.has_value() ? grad_conic.value() : torch::zeros({P, 2, 2}, means3D.options());
  Tensor dL_dopacity   = grad_opacity.has_value() ? grad_opacity.value() : torch::zeros({P, 1}, means3D.options());
  Tensor dL_dsh        = torch::zeros({P, M, 3}, means3D.options());
  Tensor dL_dscales    = torch::zeros({P, 3}, means3D.options());
  Tensor dL_drotations = torch::zeros({P, 4}, means3D.options());
  Tensor dL_dcolors    = torch::zeros({P, NUM_CHANNELS}, means3D.options());
  Tensor dL_dcov3D     = torch::zeros({P, 6}, means3D.options());
  at::optional<Tensor> dL_dextras;
  if (E > 0) dL_dextras = torch::zeros({P, E}, means3D.options());

  if (P != 0) {
    Rasterizer::backward(P, degree, M, R, E, W, H, means3D.contiguous().data<float>(), sh.contiguous().data<float>(),
        colors.contiguous().data<float>(), scales.data_ptr<float>(), scale_modifier, rotations.data_ptr<float>(),
        cov3D_precomp.contiguous().data<float>(), viewmatrix.contiguous().data<float>(),
        projmatrix.contiguous().data<float>(), campos.contiguous().data<float>(),
        E > 0 ? extras.value().contiguous().data<float>() : nullptr, tan_fovx, tan_fovy, radii.contiguous().data<int>(),
        reinterpret_cast<char*>(geomBuffer.contiguous().data_ptr()),
        reinterpret_cast<char*>(binningBuffer.contiguous().data_ptr()),
        reinterpret_cast<char*>(imageBuffer.contiguous().data_ptr()), out_opacity.contiguous().data<float>(),
        dL_dout_color.contiguous().data<float>(), dL_dout_opacity.contiguous().data<float>(),
        E > 0 ? dL_dout_extra.value().contiguous().data<float>() : nullptr, dL_dmeans2D.contiguous().data<float>(),
        dL_dconic.contiguous().data<float>(), dL_dopacity.contiguous().data<float>(),
        dL_dcolors.contiguous().data<float>(), dL_dmeans3D.contiguous().data<float>(),
        dL_dcov3D.contiguous().data<float>(), dL_dsh.contiguous().data<float>(), dL_dscales.contiguous().data<float>(),
        dL_drotations.contiguous().data<float>(), E > 0 ? dL_dextras.value().data<float>() : nullptr, debug);
  }

  return std::make_tuple(
      dL_dmeans2D, dL_dcolors, dL_dopacity, dL_dmeans3D, dL_dcov3D, dL_dsh, dL_dscales, dL_drotations, dL_dextras);
}

std::tuple<Tensor, Tensor, Tensor, Tensor, Tensor, Tensor, Tensor, Tensor> gaussian_rasterize_perpare_backward(
    // scalar parameters
    const float scale_modifier, const float tan_fovx, const float tan_fovy, const int degree,
    // tensor parameters
    const torch::Tensor& viewmatrix, const torch::Tensor& projmatrix, const torch::Tensor& campos,
    // inputs
    const torch::Tensor& means3D, const torch::Tensor& colors, const torch::Tensor& scales,
    const torch::Tensor& rotations, const torch::Tensor& cov3D_precomp, const torch::Tensor& sh,
    // outputs
    const int R, const torch::Tensor& radii, const Tensor& out_opacity,
    // grad_outputs
    const torch::Tensor& dL_dout_color, const Tensor& dL_dout_opacity,
    // grad_inputs
    torch::optional<Tensor>& grad_means2D, torch::optional<Tensor>& grad_conic, torch::optional<Tensor>& grad_opacity,
    // buffer
    const torch::Tensor& geomBuffer, const torch::Tensor& binningBuffer, const torch::Tensor& imageBuffer) {
  const int P = means3D.size(0);
  const int H = dL_dout_color.size(1);
  const int W = dL_dout_color.size(2);

  int M = 0;
  if (sh.size(0) != 0) {
    M = sh.size(1);
  }

  Tensor dL_dmeans3D   = torch::zeros({P, 3}, means3D.options());
  Tensor dL_dmeans2D   = grad_means2D.has_value() ? grad_means2D.value() : torch::zeros({P, 3}, means3D.options());
  Tensor dL_dconic     = grad_conic.has_value() ? grad_conic.value() : torch::zeros({P, 2, 2}, means3D.options());
  Tensor dL_dopacity   = grad_opacity.has_value() ? grad_opacity.value() : torch::zeros({P, 1}, means3D.options());
  Tensor dL_dcolors    = torch::zeros({P, NUM_CHANNELS}, means3D.options());
  Tensor dL_dcov3D     = torch::zeros({P, 6}, means3D.options());
  Tensor dL_dsh        = torch::zeros({P, M, 3}, means3D.options());
  Tensor dL_dscales    = torch::zeros({P, 3}, means3D.options());
  Tensor dL_drotations = torch::zeros({P, 4}, means3D.options());

  if (P != 0) {
    char* geom_buffer = reinterpret_cast<char*>(geomBuffer.contiguous().data_ptr());
    // char* binning_buffer      = reinterpret_cast<char*>(binningBuffer.contiguous().data_ptr());
    // char* img_buffer          = reinterpret_cast<char*>(imageBuffer.contiguous().data_ptr());
    GeometryState geomState = GeometryState::fromChunk(geom_buffer, P);
    // BinningState binningState = BinningState::fromChunk(binning_buffer, R);
    // ImageState imgState       = ImageState::fromChunk(img_buffer, W * H);

    const int* radii_ptr = radii.contiguous().data_ptr<int>();
    if (radii_ptr == nullptr) radii_ptr = geomState.internal_radii;
    const float focal_y = H / (2.0f * tan_fovy);
    const float focal_x = W / (2.0f * tan_fovx);

    const dim3 tile_grid((W + BLOCK_X - 1) / BLOCK_X, (H + BLOCK_Y - 1) / BLOCK_Y, 1);
    const dim3 block(BLOCK_X, BLOCK_Y, 1);
    // Take care of the rest of preprocessing.
    // Was the precomputed covariance given to us or a scales/rot pair?
    // If precomputed, pass that. If not, use the one we computed ourselves.
    const float* cov3D_ptr = cov3D_precomp.contiguous().data<float>();
    cov3D_ptr              = (cov3D_ptr != nullptr) ? cov3D_ptr : geomState.cov3D;
    preprocess_backward(P, degree, M, (float3*) means3D.contiguous().data<float>(), radii_ptr,
        sh.contiguous().data<float>(), geomState.clamped, (glm::vec3*) scales.contiguous().data<float>(),
        (glm::vec4*) rotations.data_ptr<float>(), scale_modifier, cov3D_ptr, viewmatrix.contiguous().data<float>(),
        projmatrix.contiguous().data<float>(), focal_x, focal_y, tan_fovx, tan_fovy,
        (glm::vec3*) campos.contiguous().data<float>(), (float3*) dL_dmeans2D.contiguous().data<float>(),
        dL_dconic.contiguous().data<float>(), (glm::vec3*) dL_dmeans3D.contiguous().data<float>(),
        dL_dcolors.contiguous().data<float>(), dL_dcov3D.contiguous().data<float>(), dL_dsh.contiguous().data<float>(),
        (glm::vec3*) dL_dscales.contiguous().data<float>(), (glm::vec4*) dL_drotations.contiguous().data<float>());
    CHECK_CUDA_ERROR("preprocess");
  }

  return std::make_tuple(
      dL_dmeans2D, dL_dcolors, dL_dopacity, dL_dmeans3D, dL_dcov3D, dL_dsh, dL_dscales, dL_drotations);
}

REGIST_PYTORCH_EXTENSION(
    nerf_gaussian_render_backward, { m.def("rasterize_gaussians_backward", &RasterizeGaussiansBackwardCUDA); })
}  // namespace GaussianRasterizer